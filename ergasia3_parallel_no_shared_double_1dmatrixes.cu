#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <float.h>
#include <time.h>
#include <hip/hip_runtime.h>

struct timeval startwtime,endwtime;
double seq_time;

__device__ double kernel(double a,double *d_sigma);
__device__ double distance(int l,int q,double *a, double *b, int *d_d);
void printmatrixd(int a, int b, double *matrix);
void printmatrixi(int a,int b, int *matrix);
void printsolution(int a, int b, double *point, int *kmatrix);
void checkfunction(void);

__global__ void devMain(double *d_m,int *d_kmatrix,double *d_x,double *d_y,double *d_y2, double *d_epsilon, double *d_sigma, int *d_n, int *d_d, int *d_count, double *d_xtemp, double *d_sum2);

int i,j,s,n,d,len,h;
int *d_n, *d_d;

double *sum2,*m, *xtemp;
double *d_sum2, *d_m, *d_xtemp;

int *kmatrix, *count;
int *d_kmatrix, *d_count;

FILE *myfile;

double *x, *y, *y2;
double *d_x, *d_y, *d_y2;

double sigma,epsilon,p; //only use sigma and epsilon
double *d_sigma, *d_epsilon;

int main(int argc, char **argv){
	if (argc!=5) {
		printf("Usage: %s, s, n, d, file.bin, \nwhere s is sigma*0.1, e is sigma*0.0001, \nn is the number of elements, \nd its dimension and file.bin\nthe binary file with the elements \n",argv[0]);
	    exit(1);
	}
	s=atoi(argv[1]);
	sigma=s*0.1;
	hipMalloc((void **)&d_sigma,sizeof(double));
	hipMemcpy(d_sigma,&sigma,sizeof(double),hipMemcpyHostToDevice);

	epsilon=sigma*0.0001;
	hipMalloc((void **)&d_epsilon,sizeof(double));
	hipMemcpy(d_epsilon,&epsilon,sizeof(double),hipMemcpyHostToDevice);

	n=atoi(argv[2]);
	hipMalloc((void **)&d_n,sizeof(int));
	hipMemcpy(d_n,&n,sizeof(int),hipMemcpyHostToDevice);

	d=atoi(argv[3]);
	hipMalloc((void **)&d_d,sizeof(int));
	hipMemcpy(d_d,&d,sizeof(int),hipMemcpyHostToDevice);

	xtemp=(double*)malloc(n*sizeof(double));
	hipMalloc((void **)&d_xtemp,n*sizeof(double));

	m=(double*)malloc(n*sizeof(double));
	hipMalloc((void **)&d_m,n*sizeof(double));

	sum2=(double*)malloc(n*sizeof(double));
	hipMalloc((void **)&d_sum2,n*sizeof(double));

	kmatrix=(int*)malloc(n*sizeof(int));
	hipMalloc((void **)&d_kmatrix,n*sizeof(int));

	count=(int*)malloc(n*sizeof(int));
	hipMalloc((void **)&d_count,n*sizeof(int));

	x=(double*)malloc(n*d*sizeof(double));
	hipMalloc((void **)&d_x,n*d*sizeof(double));

	y=(double*)malloc(n*d*sizeof(double));
	hipMalloc((void **)&d_y,n*d*sizeof(double));

	y2=(double*)malloc(n*d*sizeof(double));
	hipMalloc((void **)&d_y2,n*d*sizeof(double));

	for (i=0;i<n;i++){
		m[i]=100;
		sum2[i]=0;
		kmatrix[i]=0;
		xtemp[i]=0;
		count[i]=0;
	}
	hipMemcpy(d_m,m,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_sum2,sum2,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_kmatrix,kmatrix,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_xtemp,xtemp,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_count,count,n*sizeof(int),hipMemcpyHostToDevice);

	myfile=fopen(argv[4],"rb");
	for (i=0;i<n;i++){
		for (j=0;j<d;j++){
			len=fread(&p,8,1,myfile);
			x[i*d+j]=p;
			y[i*d+j]=0;
			y2[i*d+j]=p;
		}
	}
	fclose(myfile);

	hipMemcpy(d_x,x,n*d*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_y,y,n*d*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_y2,y2,n*d*sizeof(double),hipMemcpyHostToDevice);
	/** start of the main part of the program**/

	gettimeofday(&startwtime,NULL);

	devMain<<<n,1>>>(d_m,d_kmatrix,d_x,d_y,d_y2,d_epsilon,d_sigma,d_n,d_d,d_count,d_xtemp,d_sum2);

	gettimeofday(&endwtime,NULL);
	hipMemcpy(kmatrix,d_kmatrix,n*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(y2,d_y2,n*d*sizeof(double),hipMemcpyDeviceToHost);

	printsolution(n,d,y2,kmatrix);
	//checkfunction();

	seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
		      + endwtime.tv_sec - startwtime.tv_sec);
	printf("total time: %f secs\n",seq_time);

    free(x);
    free(y);
    free(y2);
    free(count);
    free(kmatrix);
    //free (sum);
    free (sum2);
    free(m);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_y2);
    hipFree(d_count);
    hipFree(d_kmatrix);
    hipFree(d_sum2);
    //cudaFree(d_sum);
    hipFree(m);  
}

__device__ double kernel(double a, double *d_sigma){
	double result;
	result=exp(-a/(2*(*d_sigma)*(*d_sigma)));
	return result;
}

__device__ double distance(int l,int q, double *a, double *b, int *d_d){
	double sum=0;
	double root=0;
	int j;
	for (j=0;j<(*d_d);j++){
		sum+=(a[l*(*d_d)+j]-b[q*(*d_d)+j])*(a[l*(*d_d)+j]-b[q*(*d_d)+j]);
	}
	root=sqrt(sum);
	return root;
}

void printmatrixd(int a,int b, double *matrix){
	int i,j;
	for (i=0;i<a;i++){
		for (j=0;j<b;j++){
			printf("%f ",matrix[i*b+j]);
		}
		printf("\n");
	}
}

void printmatrixi(int a,int b, int *matrix){
	int i,j;
	for (i=0;i<a;i++){
		for (j=0;j<b;j++){
			printf("%d ",matrix[i*b+j]);
		}
		printf("\n");
	}
}

void printsolution(int a, int b, double *point, int *kmatrix){
	int i,j;
	for (i=0;i<a;i++){
		for (j=0;j<b;j++){
			printf("%f ",point[i*b+j]);
		}
		printf("i=%d, k=%d\n",i, kmatrix[i]);
	}
}

void checkfunction(void){
	char binary[50];
	FILE *myfile2;
	int  count,i,j;
	double *a;
	double p,dist;
	a=(double*)malloc(n*d*sizeof(double));
	printf("please give the name of the binary file \n(of double floats-8-bytes) that is a right solution\nto the mean shift problem:\n");
	dist=0;
	count=0;

	scanf("%s",binary);
	myfile2=fopen(binary,"rb");
	for (i=0;i<n;i++){
		//printf("reached here\n");
		for (int j=0;j<d;j++){
			fread(&p,8,1,myfile);
			a[i*d+j]=p;
		}
	}
	fclose(myfile2);
	for (i=0;i<n;i++){
		for (j=0;j<d;j++){
			dist+=(a[i*d+j]-y2[i*d+j])*(a[i*d+j]-y2[i*d+j]);
		}
		if (dist>sigma*sigma/100){ //distance>sigma/10
			count++;
		}
		dist=0;
	}
	free(a);
	printf("we have problem in %d out of %d points\n",count,n);



}

__global__ void devMain(double *d_m,int *d_kmatrix,double *d_x,double *d_y,double *d_y2, double *d_epsilon, double *d_sigma, int *d_n, int *d_d, int *d_count, double *d_xtemp, double *d_sum2){
	int j;
	int h;

		while ((d_m[blockIdx.x]>*d_epsilon)&&(d_kmatrix[blockIdx.x]<15)){
			d_count[blockIdx.x]=0;
			for (j=0;j<*d_n;j++){
				//printf("j=%d\n",j);
				if (distance(blockIdx.x,j,d_y2,d_x,d_d)<(*d_sigma)*(*d_sigma)){
					d_xtemp[blockIdx.x]=kernel(distance(blockIdx.x,j,d_y2,d_x,d_d),d_sigma);
					d_count[blockIdx.x]++;
					d_sum2[blockIdx.x]+=d_xtemp[blockIdx.x];
					for (h=0;h<*d_d;h++){
						d_y[blockIdx.x*(*d_d)+h]+=d_xtemp[blockIdx.x]*d_x[j*(*d_d)+h];
					}
				}
			}
			for (h=0;h<*d_d;h++){
				d_y[blockIdx.x*(*d_d)+h]/=d_sum2[blockIdx.x];
			}
			d_sum2[blockIdx.x]=0;
			d_m[blockIdx.x]=distance(blockIdx.x,blockIdx.x,d_y,d_y2,d_d);
			d_kmatrix[blockIdx.x]++;
			for (h=0;h<(*d_d);h++){
				d_y2[blockIdx.x*(*d_d)+h]=d_y[blockIdx.x*(*d_d)+h];
				d_y[blockIdx.x*(*d_d)+h]=0;
			}
		}
	
}