#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <float.h>
#include <time.h>
#include <hip/hip_runtime.h>

struct timeval startwtime,endwtime;
double seq_time;

__device__ double kernel(double a,double *d_sigma);
__device__ double distance(int l,int q,double *a, double *b, int *d_d);
void printmatrixd(int a, int b, double *matrix);
void printmatrixi(int a,int b, int *matrix);
void printsolution(int a, int b, double *point, int *kmatrix);
void checkfunction(void);

__global__ void devMain(double *d_m,int *d_kmatrix,double *d_x,double *d_y,double *d_y2, double *d_epsilon, double *d_sigma, int *d_n, int *d_d, int *d_count, double *d_xtemp, double *d_sum2, int *d_numthreads, double *d_sum);

int i,j,s,n,d,len,h,numthreads;
int *d_n, *d_d;

double *sum2,*m, *xtemp, *sum;
double *d_sum2, *d_m, *d_xtemp, *d_sum;

int *kmatrix, *count;
int *d_kmatrix, *d_count, *d_numthreads;

FILE *myfile;

double *x, *y, *y2;
double *d_x, *d_y, *d_y2;

double sigma,epsilon,p; //only use sigma and epsilon
double *d_sigma, *d_epsilon;

int main(int argc, char **argv){
	numthreads=6 ;// you can change this if you want
	if (argc!=5) {
		printf("Usage: %s, s, n, d, file.bin, \nwhere s is sigma*0.1, e is sigma*0.0001, \nn is the number of elements, \nd its dimension and file.bin\nthe binary file with the elements \n",argv[0]);
	    exit(1);
	}
	s=atoi(argv[1]);
	sigma=s*0.1;
	hipMalloc((void **)&d_sigma,sizeof(double));
	hipMemcpy(d_sigma,&sigma,sizeof(double),hipMemcpyHostToDevice);

	
	hipMalloc((void **)&d_numthreads,sizeof(int));
	hipMemcpy(d_numthreads,&numthreads,sizeof(int),hipMemcpyHostToDevice);



	epsilon=sigma*0.0001;
	hipMalloc((void **)&d_epsilon,sizeof(double));
	hipMemcpy(d_epsilon,&epsilon,sizeof(double),hipMemcpyHostToDevice);

	n=atoi(argv[2]);
	hipMalloc((void **)&d_n,sizeof(int));
	hipMemcpy(d_n,&n,sizeof(int),hipMemcpyHostToDevice);

	d=atoi(argv[3]);
	hipMalloc((void **)&d_d,sizeof(int));
	hipMemcpy(d_d,&d,sizeof(int),hipMemcpyHostToDevice);

	xtemp=(double*)malloc(n*numthreads*sizeof(double));
	hipMalloc((void **)&d_xtemp,n*numthreads*sizeof(double));

	m=(double*)malloc(n*sizeof(double));
	hipMalloc((void **)&d_m,n*sizeof(double));

	sum2=(double*)malloc(n*numthreads*sizeof(double));
	hipMalloc((void **)&d_sum2,n*numthreads*sizeof(double));

	sum=(double*)malloc(n*sizeof(double));
	hipMalloc((void **)&d_sum,n*sizeof(double));

	kmatrix=(int*)malloc(n*sizeof(int));
	hipMalloc((void **)&d_kmatrix,n*sizeof(int));

	count=(int*)malloc(n*sizeof(int));
	hipMalloc((void **)&d_count,n*sizeof(int));

	x=(double*)malloc(n*d*sizeof(double));
	hipMalloc((void **)&d_x,n*d*sizeof(double));

	y=(double*)malloc(n*d*sizeof(double));
	hipMalloc((void **)&d_y,n*d*sizeof(double));

	y2=(double*)malloc(n*d*sizeof(double));
	hipMalloc((void **)&d_y2,n*d*sizeof(double));

	for (i=0;i<n;i++){
		m[i]=100;
		for (j=0;j<numthreads;j++){
			xtemp[i*numthreads+j]=0;
			sum2[i*numthreads+j]=0;
		}
		sum[i]=0;
		kmatrix[i]=0;
		count[i]=0;
	}
	hipMemcpy(d_m,m,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_sum,sum,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_sum2,sum2,n*numthreads*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_kmatrix,kmatrix,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_xtemp,xtemp,n*numthreads*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_count,count,n*sizeof(int),hipMemcpyHostToDevice);

	myfile=fopen(argv[4],"rb");
	for (i=0;i<n;i++){
		for (j=0;j<d;j++){
			len=fread(&p,8,1,myfile);
			x[i*d+j]=p;
			y[i*d+j]=0;
			y2[i*d+j]=p;
		}
	}
	fclose(myfile);

	hipMemcpy(d_x,x,n*d*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_y,y,n*d*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_y2,y2,n*d*sizeof(double),hipMemcpyHostToDevice);
	/** start of the main part of the program**/

	gettimeofday(&startwtime,NULL);

	devMain<<<n,numthreads,n*d*sizeof(double)>>>(d_m,d_kmatrix,d_x,d_y,d_y2,d_epsilon,d_sigma,d_n,d_d,d_count,d_xtemp,d_sum2,d_numthreads,d_sum);

	gettimeofday(&endwtime,NULL);
	hipMemcpy(kmatrix,d_kmatrix,n*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(y2,d_y2,n*d*sizeof(double),hipMemcpyDeviceToHost);

	printsolution(n,d,y2,kmatrix);
	//checkfunction();

	seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
		      + endwtime.tv_sec - startwtime.tv_sec);
	printf("total time: %f secs\n",seq_time);

    free(x);
    free(y);
    free(y2);
    free(count);
    free(kmatrix);
    free (sum);
    free (sum2);
    free(m);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_y2);
    hipFree(d_count);
    hipFree(d_kmatrix);
    hipFree(d_sum2);
    hipFree(d_sum);
    hipFree(m);

}

__device__ double kernel(double a, double *d_sigma){
	double result;
	result=exp(-a/(2*(*d_sigma)*(*d_sigma)));
	return result;
}

__device__ double distance(int l,int q, double *a, double *b, int *d_d){
	double sum=0;
	double root=0;
	int j;
	for (j=0;j<(*d_d);j++){
		sum+=(a[l*(*d_d)+j]-b[q*(*d_d)+j])*(a[l*(*d_d)+j]-b[q*(*d_d)+j]);
	}
	root=sqrt(sum);
	return root;
}

void printmatrixd(int a,int b, double *matrix){
	int i,j;
	for (i=0;i<a;i++){
		for (j=0;j<b;j++){
			printf("%f ",matrix[i*b+j]);
		}
		printf("\n");
	}
}

void printmatrixi(int a,int b, int *matrix){
	int i,j;
	for (i=0;i<a;i++){
		for (j=0;j<b;j++){
			printf("%d ",matrix[i*b+j]);
		}
		printf("\n");
	}
}

void printsolution(int a, int b, double *point, int *kmatrix){
	int i,j;
	for (i=0;i<a;i++){
		for (j=0;j<b;j++){
			printf("%f ",point[i*b+j]);
		}
		printf("i=%d, k=%d\n",i, kmatrix[i]);
	}
}

void checkfunction(void){
	char binary[50];
	FILE *myfile2;
	int  count,i,j;
	double *a;
	double p,dist;
	a=(double*)malloc(n*d*sizeof(double));
	printf("please give the name of the binary file \n(of double floats-8-bytes) that is a right solution\nto the mean shift problem:\n");
	dist=0;
	count=0;

	scanf("%s",binary);
	myfile2=fopen(binary,"rb");
	for (i=0;i<n;i++){
		//printf("reached here\n");
		for (int j=0;j<d;j++){
			fread(&p,8,1,myfile);
			a[i*d+j]=p;
		}
	}
	fclose(myfile2);
	for (i=0;i<n;i++){
		for (j=0;j<d;j++){
			dist+=(a[i*d+j]-y2[i*d+j])*(a[i*d+j]-y2[i*d+j]);
		}
		if (dist>sigma*sigma/100){ //distance>sigma/10
			count++;
		}
		dist=0;
	}
	free(a);
	printf("we have problem in %d out of %d points\n",count,n);



}

__global__ void devMain(double *d_m,int *d_kmatrix,double *d_x,double *d_y,double *d_y2, double *d_epsilon, double *d_sigma, int *d_n, int *d_d, int *d_count, double *d_xtemp, double *d_sum2, int *d_numthreads, double *d_sum){
	int j,i;
	int h;
	int bid=blockIdx.x;
	int tid=threadIdx.x;
	int steps=(*d_n/(*d_numthreads));
	extern __shared__ double sharedx[];
	

	for (j=0;j<(*d_d);j++){
		sharedx[bid*(*d_d)+j]=d_x[bid*(*d_d)+j];
	}
	__syncthreads();

		while ((d_m[bid]>*d_epsilon)&&(d_kmatrix[bid]<15)){
			//d_count[bid]=0;
			for (j=tid*steps;j<(tid+1)*steps;j++){
				//printf("j=%d\n",j);
				if (distance(bid,j,d_y2,d_x,d_d)<(*d_sigma)*(*d_sigma)){
					d_xtemp[bid*(*d_numthreads)+tid]=kernel(distance(bid,j,d_y2,d_x,d_d),d_sigma);
					//d_count[bid*(*d_numthreads)+tid]++;
					d_sum2[bid*(*d_numthreads)+tid]+=d_xtemp[bid*(*d_numthreads)+tid];
					for (h=0;h<(*d_d);h++){
						for (i=0;i<(*d_numthreads);i++){
						if (i==tid)	//to prevent data races
						d_y[bid*(*d_d)+h]+=d_xtemp[bid*(*d_numthreads)+tid]*d_x[j*(*d_d)+h];
					    __syncthreads();
					}
					}
				}
			}
			if (1==tid){
			for (h=0;h<(*d_numthreads);h++){
				d_sum[bid]+=d_sum2[bid*(*d_numthreads)+h];
			}
			for (h=0;h<*d_d;h++){
				d_y[bid*(*d_d)+h]/=d_sum[bid];
			}
			d_sum[bid]=0;
			d_m[bid]=distance(bid,bid,d_y,d_y2,d_d);
			d_kmatrix[bid]++;
			for (h=0;h<(*d_d);h++){
				d_y2[bid*(*d_d)+h]=d_y[bid*(*d_d)+h];
				d_y[bid*(*d_d)+h]=0;
			}
		}
		__syncthreads();
		d_sum2[bid*(*d_numthreads)+tid]=0;
		__syncthreads();
	}
	
}